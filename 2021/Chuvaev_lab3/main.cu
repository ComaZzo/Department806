#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctime>

#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define E 10e-4
#define BLOCKS 32
#define THREADS 128
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__device__ __host__ float func(float x){
    return log(3.66*x) - 4.12 * x + 1.5;
}


__device__ __host__ float func_derivative(float x){
    return 1 / (3.66*x) - 4.12;
}


__device__ __host__ float func_derivative_second(float x){
    return -1 / (3.66*x*x);
}


__global__ void newton_method(float *c, double step, float A){   
    float a = A + blockIdx.x * threadIdx.x * step;
    float b = A + (blockIdx.x * threadIdx.x + 1) * step;
    if (func(a) * func(b) > 0)
        return;
    double calc;
    do{
        calc = calc - func(calc) / func_derivative(calc);
    }while (fabs(func(calc)) >= E);
    c[blockIdx.x * threadIdx.x] = calc;
}


void cpu_execute(){
    int start = clock(), time;
    double c=2; 
    int n=0;
    while (fabs(func(c))>=E)
    {
        c = c-func(c) / func_derivative(c);
        n++;
    }
    time = clock() - start;
 	printf("==============================   CPU TIME   ===============================\n");
    printf("Equation root = %lf\n",c);
    printf("Iteration number: n = %d\n",n); 
 	printf("\nCPU compute time: %.5f microseconds\n\n", time*1000);
}

void gpu_execute(){
    float B = 10, A = 0.01; // common borders

    hipEvent_t start, stop;
	float gpuTime = 0.0f;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    const int n = THREADS * BLOCKS;
    float step = fabs(A - B) / n;
    // float c[n];
    // float* dev_c;
    thrust::host_vector<float> c (n);
    thrust::device_vector<float> dev_c (n);

    // hipMalloc(&dev_c, n * sizeof(float));

    newton_method <<<BLOCKS, THREADS >>> (thrust::raw_pointer_cast(dev_c.data()), step, A);
    // hipMemcpy(c, 
    //                         dev_c, 
    //                         n * sizeof(unsigned int), 
    //                         hipMemcpyDeviceToHost); 
    c = dev_c;

    HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
  	HANDLE_ERROR(hipEventElapsedTime(&gpuTime, start, stop));
    printf("==============================   GPU TIME   ===============================\n");
 	printf("\nGPU compute time: %.5f microseconds\n\n", gpuTime);

    for (unsigned int i = 0; i < c.size(); i++)
    {
        if (c[i] > E)
        {
            printf("GPU root %f \n", c[i]);
        }
    }
    // hipFree(dev_c);
    // hipDeviceReset();
}

int main(void){
    gpu_execute();
    cpu_execute();
    return 0;
}