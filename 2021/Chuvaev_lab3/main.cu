
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <cmath>


#define E 10e-4
#define BLOCKS 32
#define THREADS 128


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__device__ __host__ float func(float x){
    return log(3.66*x) - 4.12 * x + 1.5;
}


__device__ __host__ float func_derivative(float x){
    return 1 / (3.66*x) - 4.12;
}


__device__ __host__ float func_derivative_second(float x){
    return -1 / (3.66*x*x);
}

__global__ void newton_method(float *c, double step){
    float A = 0;    
    float a = A + blockIdx.x * threadIdx.x * step;
    float b = A + (blockIdx.x * threadIdx.x + 1) * step;
    if (func(a) * func(b) > 0)
        return;
    double calc;
    do{
        calc = calc - func(calc) / func_derivative(calc);
    }while (fabs(func(calc)) >= E);
    c[blockIdx.x * threadIdx.x] = calc;
}

void cpu_execute(){
    int start = clock(), time;
    double c=2; 
    int n=0;
    while (fabs(func(c))>=E)
    {
        c=c-func(c) / func_derivative(c);
        n++;
    }
    time = clock() - start;
 	printf("==============================   CPU TIME   ===============================\n");
    printf("Equation root = %lf\n",c);
    printf("Iteration number: n = %d\n",n); 
 	printf("\nCPU compute time: %.5f microseconds\n\n", time*1000);
}

void gpu_execute(){
    float B = 10, A = 0; // common borders

    hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start, 0);

    const int n = THREADS * BLOCKS;
    float c[n];
    float step = fabs(A - B) / n;
    float* dev_c;
    HANDLE_ERROR(hipMalloc(&dev_c, 
                            n * sizeof(float)));

    newton_method <<<BLOCKS, THREADS >>> (dev_c, step);

    HANDLE_ERROR(hipMemcpy(c, 
                            dev_c, 
                            n * sizeof(unsigned int), 
                            hipMemcpyDeviceToHost));
    
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
  	hipEventElapsedTime(&gpuTime, start, stop);
    printf("==============================   GPU TIME   ===============================\n");
 	printf("\nGPU compute time: %.5f microseconds\n\n", gpuTime);

    for (unsigned int i = 0; i < n; i++)
    {
        if (c[i] > E)
        {
            printf("GPU root %f \n", c[i]);
        }
    }
    hipFree(dev_c);
    hipDeviceReset();
}

int main(void){
    gpu_execute();
    cpu_execute();
    return 0;
}