﻿

//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define SIZE	16

__global__ void Square(int* a, int* c, int n) //Добавляем __global__ чтобы функция выполнялась на GPU(многопоточно)
{
	int i = threadIdx.x; //Задаем как поточную переменную
	if (i<n ) //Проверка наличия памяти для записи переменной
		c[i] = a[i] * a[i];
}

int main()
{
	float elapsedTime;
	clock_t start = clock();

	int *a, *c;	//Инициализация переменных
	int *d_a, *d_c;

	a = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));

	hipMalloc(&d_a, SIZE * sizeof(int)); //Выделение памяти на переменную
	hipMalloc(&d_c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		c[i] = 0;
	}

	hipMemcpy( d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice); //Копируем данные в GPU из CPU
	hipMemcpy( d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	Square<<< 1, SIZE >>>(d_a, d_c, SIZE); //Вызов функции со специальной конфигурацией( <<< *кол-во блоков*, *величина блока*>>>...)

	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost); //Копируем данные обратно из GPU в CPU для вывода


	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	free(a);
	free(c);

	hipFree(d_a);
	hipFree(d_c); //Освобождаем память

	elapsedTime = ((double)clock() - start) / CLOCKS_PER_SEC; // Подсчет времени
	printf("GPU time elapsed: %f seconds \n", elapsedTime); 

	return 0;
}