#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 16

#define BLOCK_SIZE 16

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> aTex;
texture<float, 2, hipReadModeElementType> bTex;

__global__ void matMult(int n, float * c)
{
	int   bx = blockIdx.x;
	int   by = blockIdx.y;
	int   tx = threadIdx.x;
	int   ty = threadIdx.y;
	float sum = 0.0f;
	int   ia = n * BLOCK_SIZE * by + n * ty;
	int   ib = BLOCK_SIZE * bx + tx;
	int   ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	for (int k = 0; k < n; k++)
		//sum += d_a[ia + k] * d_b[ib + k*n];
		sum += tex2D(aTex, ia, k)*tex2D(bTex, ib, k*n);

	c[ic + n * ty + tx] = sum;
}


int main(int argc, char *  argv[])
{
	int         numBytes = N * N * sizeof (float);
	hipArray	* adev, *bdev;
	float       *cdev;
	dim3        threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3        blocks(N / threads.x, N / threads.y);

	//Generate matricies
	float * h_a = new float[N*N];
	float * h_b = new float[N*N];
	float * c = new float[N*N];

	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			h_a[i*N + j] = 1;
			h_b[i*N + j] = 2;
		}
	}

	unsigned int size = N * N * sizeof(float);
	// Allocate array and copy data
	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipMallocArray(&adev,
		&channelDesc,
		N,
		N);
	hipMemcpyToArray(adev,
		0,
		0,
		h_a,
		size,
		hipMemcpyHostToDevice);

	hipMallocArray(&bdev,
		&channelDesc,
		N,
		N);
	hipMemcpyToArray(bdev,
		0,
		0,
		h_b,
		size,
		hipMemcpyHostToDevice);

	// Bind the array to the texture
	hipBindTextureToArray(aTex, adev, channelDesc);
	hipBindTextureToArray(bTex, bdev, channelDesc);


	hipMalloc((void**)&cdev, numBytes);	// allocate DRAM

	//hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4))


	matMult << <blocks, threads >> > (N, cdev);

	hipDeviceSynchronize();
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%.5f ", c[i*N + j]);
		}
		printf("\n");
	}

	//hipUnbindTexture(aTex);
	//hipUnbindTexture(bTex);
	// free GPU memory
	//hipFreeArray(adev);
	//hipFreeArray(adev);
	hipFree(cdev);

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();

	return 0;
}
