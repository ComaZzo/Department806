#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "math.h"

// includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization

#define	N	(1024*1024)		

__global__ void kernel(float * data)
{
	int   idx = blockIdx.x * blockDim.x + threadIdx.x;
	float x = 2.0f * 3.1415926f * (float)idx / (float)N;

	data[idx] = sinf(sqrtf(x));
}

int main(int argc, char *  argv[])
{
	float * a = new float [N];
	
	float elapsedTimeInMsGPU = 0.0f;
	float elapsedTimeInMsCPU = 0.0f;
	StopWatchInterface *timer = NULL;


	//GPU restart
	hipDeviceReset();

	//Entry point to mesure time
	hipEvent_t start, stop;
	//GPU timer
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	//SDK timer
	sdkCreateTimer(&timer);

	//Start the timer
	checkCudaErrors(hipEventRecord(start, 0));
	sdkStartTimer(&timer);

	float * dev = NULL;
	
	hipMalloc((void**)&dev, N * sizeof (float));

	kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);

	hipMemcpy(a, dev, N * sizeof (float), hipMemcpyDeviceToHost);
	hipFree(dev);

	//Stop the timer
	checkCudaErrors(hipEventRecord(stop, 0));
	sdkStopTimer(&timer);

	// make sure GPU has finished copying
	checkCudaErrors(hipDeviceSynchronize());
	
	//Finish point to mesure time
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMsGPU, start, stop));
	elapsedTimeInMsCPU = sdkGetTimerValue(&timer);

	printf("Execution time in ms via CPU timer %f\n", elapsedTimeInMsCPU);
	printf("Execution time in ms via GPU timer %f\n", elapsedTimeInMsGPU);
	//for (int idx = 0; idx < N; idx++)  printf("a[%d] = %.5f\n", idx, a[idx]);

	return 0;
}
