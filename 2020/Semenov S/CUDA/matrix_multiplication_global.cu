#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// includes
#include "helper_functions.h"  // helper for shared functions common to CUDA Samples
#include "hip/hip_runtime_api.h"       // helper functions for CUDA error checking and initialization

#define N 1024

#define BLOCK_SIZE 16

__global__ void matMult(float * a, float * b, int n, float * c)
{
	int   bx = blockIdx.x;
	int   by = blockIdx.y;
	int   tx = threadIdx.x;
	int   ty = threadIdx.y;
	float sum = 0.0f;
	int   ia = n * BLOCK_SIZE * by + n * ty;
	int   ib = BLOCK_SIZE * bx + tx;
	int   ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	for (int k = 0; k < n; k++)
		sum += a[ia + k] * b[ib + k*n];

	c[ic + n * ty + tx] = sum;
}


int main(int argc, char *  argv[])
{
	int         numBytes = N * N * sizeof (float);
	float     * adev, *bdev, *cdev;
	dim3        threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3        blocks(N / threads.x, N / threads.y);

	//Generate matricies
	float * a = new float[N*N];
	float * b = new float[N*N];
	float * c = new float[N*N];

	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			a[i*N+j] = 1;
			b[i*N + j] = 2;
		}
	}

	hipMalloc((void**)&adev, numBytes);	// allocate DRAM
	hipMalloc((void**)&bdev, numBytes);	// allocate DRAM
	hipMalloc((void**)&cdev, numBytes);	// allocate DRAM
	// copy from CPU to DRAM
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

	matMult <<<blocks, threads >>> (adev, bdev, N, cdev);

	hipDeviceSynchronize();
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

	/*for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%.5f ", c[i*N + j]);
		}
		printf("\n");
	}*/

	// free GPU memory
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();

	return 0;
}
