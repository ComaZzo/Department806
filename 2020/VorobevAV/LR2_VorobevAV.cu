#include "hip/hip_runtime.h"

#include <stdio.h>
#include <ctime>
#include <cmath>

#define N (1024)

__global__ void kernel(float *dev)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (N % idx == 0) {
    dev[idx] = (float) idx;
    }
}

int main (int argc, char *argv[])
{
//------------------— GPU PART —----------------—
    float arr [N];
    float *dev = NULL;

    hipMalloc(&dev, N * sizeof(float));

    kernel<<<2, 512>>> (dev);

    hipMemcpy(&arr, dev, N * sizeof(float), hipMemcpyDeviceToHost);   

    for (int idx = 0; idx < N; idx++) 
    {
        if (arr[idx] != 0) {
            printf("%f ", arr[idx]);
        } 
    }
 
    hipFree(dev);

    return 0;
}